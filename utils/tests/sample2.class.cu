class DGLM3D {    
public:
    hipMalloc(&WDr,  NP*NP*sizeof(REAL));
    hipMalloc(&LIFT, NP*NFACE*NFP*sizeof(REAL));
    hipMalloc(&rx,   nelem*sizeof(REAL));
    hipMalloc(&vmapP, nelem*NFACE*NFP*sizeof(int));
    hipMalloc(&maxv, (nelem*NFACE + buf_size)*sizeof(REAL));
    
    ...
    
    for (int i=0; i<NVAR; i++) {
        hipMalloc(&p_u[i],      (nelem*NP + buf_size*NFP)*sizeof(REAL));
        hipMalloc(&p_ub[i],     nelem*NFACE*NFP*sizeof(REAL));        
        hipMalloc(&p_fluxLM[i], nelem*NFACE*NFP*sizeof(REAL));
    }    
};
