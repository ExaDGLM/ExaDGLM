#include <vector>
#include <string>
#include <hip/hip_runtime.h>
#include "common.h"

using namespace std;


class DataDev {    
public:
    // DG operator matrix
    REAL *WDr, *WDs, *WDt, *LIFT;
    
    // mesh coordinate transformations
    REAL *rx, *ry, *rz, *sx, *sy, *sz, *tx, *ty, *tz;
    REAL *Fscale;
    
    // mesh normal vectors
    REAL *nx, *ny, *nz;
    
    // mesh physical coordinates
    REAL *px, *py, *pz;
    
    // DG parameters
    REAL *tau, *maxv, *maxv_fs;
    
    // mesh index arrays
    int *EtoB, *vmapF, *vmapP, *Fmask;
    
    // MPI communication index arrays
    int *sendbuf_face_idxs, *sendbuf_nfp_idxs;
    REAL *sendbuf_face, *sendbuf_nfp;
    REAL *recvbuf_face, *recvbuf_nfp;
    
    // copy device to host
    vector<REAL> u;
    
    // Pack of CUDA array pointers for cublasTgemmBatched
    // Note that they require both host vectors and device pointers
    vector<REAL*> p_WDr, p_WDs, p_WDt, p_LIFT;
    vector<REAL*> p_u, p_k, p_utmp, p_ub;
    vector<REAL*> p_fluxR, p_fluxS, p_fluxT, p_fluxLM; 
    REAL **pd_WDr, **pd_WDs, **pd_WDt, **pd_LIFT;
    REAL **pd_u, **pd_k, **pd_utmp, **pd_ub;
    REAL **pd_fluxR, **pd_fluxS, **pd_fluxT, **pd_fluxLM;
    
    int nelem;
    int buf_size;
    DataHost host;
    
    
    DataDev(DataHost &host) : host(host) {
        nelem = host.nelem;
        buf_size = host.buf_size;
        
        // vector arrays
        u.resize(nelem*NP);
            
        // CUDA arrays
        alloc_dev_arrays();
        alloc_dev_pack_arrays();
        init_dev_arrays();
        
        // set the shared memory bank size to 4(float) or 8(double) bytes
        hipSharedMemConfig config = cudaSharedMemBankSizeTByte;
        hipDeviceSetSharedMemConfig(config);        
        check_cuda_error("copy_h2d_all_shared_mem");
    }
    
    ~DataDev() {
        free_dev_arrays();
    }

    void check_cuda_error(string funcname) {
        hipError_t err = hipGetLastError();
        if (err != hipSuccess)
            cout << "CUDA error(" << funcname << "): " << hipGetErrorString(err) << endl;
    }    
    
    void alloc_dev_arrays() {
        hipMalloc(&WDr,  pNP*NP*sizeof(REAL));
        hipMalloc(&WDs,  pNP*NP*sizeof(REAL));
        hipMalloc(&WDt,  pNP*NP*sizeof(REAL));
        hipMalloc(&LIFT, pNP*NFACE*NFP*sizeof(REAL));
        hipMalloc(&rx,     nelem*sizeof(REAL));
        hipMalloc(&ry,     nelem*sizeof(REAL));
        hipMalloc(&rz,     nelem*sizeof(REAL));
        hipMalloc(&sx,     nelem*sizeof(REAL));
        hipMalloc(&sy,     nelem*sizeof(REAL));
        hipMalloc(&sz,     nelem*sizeof(REAL));
        hipMalloc(&tx,     nelem*sizeof(REAL));        
        hipMalloc(&ty,     nelem*sizeof(REAL));        
        hipMalloc(&tz,     nelem*sizeof(REAL));
        hipMalloc(&Fscale, nelem*NFACE*sizeof(REAL));
        hipMalloc(&nx,     nelem*NFACE*sizeof(REAL));
        hipMalloc(&ny,     nelem*NFACE*sizeof(REAL));
        hipMalloc(&nz,     nelem*NFACE*sizeof(REAL));
        hipMalloc(&px,     nelem*NP*sizeof(REAL));
        hipMalloc(&py,     nelem*NP*sizeof(REAL));
        hipMalloc(&pz,     nelem*NP*sizeof(REAL));        
        hipMalloc(&EtoB,   nelem*NFACE*sizeof(int));
        hipMalloc(&vmapF,  nelem*NFACE*sizeof(int));
        hipMalloc(&vmapP,  nelem*NFACE*NFP*sizeof(int));
        hipMalloc(&Fmask,  NFACE*NFP*sizeof(int));        
        hipMalloc(&tau,     nelem*NFACE*sizeof(REAL));
        hipMalloc(&maxv,    (nelem*NFACE + buf_size)*sizeof(REAL));
        hipMalloc(&maxv_fs, nelem*NFACE*sizeof(REAL));
        hipMalloc(&sendbuf_face_idxs, buf_size*sizeof(int));
        hipMalloc(&sendbuf_nfp_idxs,  buf_size*NFP*sizeof(int));
        hipMalloc(&sendbuf_face, buf_size*sizeof(REAL));
        hipMalloc(&recvbuf_face, buf_size*sizeof(REAL));
        hipMalloc(&sendbuf_nfp,  buf_size*NFP*NVAR*sizeof(REAL));
        hipMalloc(&recvbuf_nfp,  buf_size*NFP*NVAR*sizeof(REAL));
        
        check_cuda_error("alloc_dev_arrays");
    }
    
    void alloc_dev_pack_arrays() {
        // C++ vector arrays are used for saving the pointer of CUDA arrays
        p_WDr.resize(NVAR);
        p_WDs.resize(NVAR);
        p_WDt.resize(NVAR);
        p_LIFT.resize(NVAR);
        p_u.resize(NVAR);
        p_k.resize(NVAR);
        p_utmp.resize(NVAR);
        p_ub.resize(NVAR);
        p_fluxR.resize(NVAR);
        p_fluxS.resize(NVAR);
        p_fluxT.resize(NVAR);
        p_fluxLM.resize(NVAR);
                
        for (int i=0; i<NVAR; i++) {
            hipMalloc(&p_u[i],      (nelem*NP + buf_size*NFP)*sizeof(REAL));
            hipMalloc(&p_k[i],      nelem*NP*sizeof(REAL));
            hipMalloc(&p_ub[i],     nelem*NFACE*NFP*sizeof(REAL));
            hipMalloc(&p_utmp[i],   nelem*pNP*sizeof(REAL));
            hipMalloc(&p_fluxR[i],  nelem*pNP*sizeof(REAL));
            hipMalloc(&p_fluxS[i],  nelem*pNP*sizeof(REAL));
            hipMalloc(&p_fluxT[i],  nelem*pNP*sizeof(REAL));
            hipMalloc(&p_fluxLM[i], nelem*pNFF*sizeof(REAL));
            hipMemset(p_k[i], 0, nelem*NP*sizeof(REAL));  // zero initialize
        }        
        
        // Pack of CUDA array pointers
        hipMalloc((void**)&pd_WDr,    NVAR*sizeof(REAL*));
        hipMalloc((void**)&pd_WDs,    NVAR*sizeof(REAL*));
        hipMalloc((void**)&pd_WDt,    NVAR*sizeof(REAL*));
        hipMalloc((void**)&pd_LIFT,   NVAR*sizeof(REAL*));
        hipMalloc((void**)&pd_u,      NVAR*sizeof(REAL*));
        hipMalloc((void**)&pd_k,      NVAR*sizeof(REAL*));
        hipMalloc((void**)&pd_utmp,   NVAR*sizeof(REAL*));
        hipMalloc((void**)&pd_ub,     NVAR*sizeof(REAL*));
        hipMalloc((void**)&pd_fluxR,  NVAR*sizeof(REAL*));
        hipMalloc((void**)&pd_fluxS,  NVAR*sizeof(REAL*));
        hipMalloc((void**)&pd_fluxT,  NVAR*sizeof(REAL*));
        hipMalloc((void**)&pd_fluxLM, NVAR*sizeof(REAL*));
        
        // copy pointers
        p_WDr  = {WDr,  WDr,  WDr,  WDr,  WDr};
        p_WDs  = {WDs,  WDs,  WDs,  WDs,  WDs};
        p_WDt  = {WDt,  WDt,  WDt,  WDt,  WDt};
        p_LIFT = {LIFT, LIFT, LIFT, LIFT, LIFT};        
        
        constexpr auto h2d = hipMemcpyHostToDevice;
        hipMemcpy(pd_WDr,    p_WDr.data(),    NVAR*sizeof(REAL*), h2d);
        hipMemcpy(pd_WDs,    p_WDs.data(),    NVAR*sizeof(REAL*), h2d);
        hipMemcpy(pd_WDt,    p_WDt.data(),    NVAR*sizeof(REAL*), h2d);
        hipMemcpy(pd_LIFT,   p_LIFT.data(),   NVAR*sizeof(REAL*), h2d);
        hipMemcpy(pd_u,      p_u.data(),      NVAR*sizeof(REAL*), h2d);
        hipMemcpy(pd_k,      p_k.data(),      NVAR*sizeof(REAL*), h2d);
        hipMemcpy(pd_utmp,   p_utmp.data(),   NVAR*sizeof(REAL*), h2d);
        hipMemcpy(pd_ub,     p_ub.data(),     NVAR*sizeof(REAL*), h2d);        
        hipMemcpy(pd_fluxR,  p_fluxR.data(),  NVAR*sizeof(REAL*), h2d);
        hipMemcpy(pd_fluxS,  p_fluxS.data(),  NVAR*sizeof(REAL*), h2d);
        hipMemcpy(pd_fluxT,  p_fluxT.data(),  NVAR*sizeof(REAL*), h2d);
        hipMemcpy(pd_fluxLM, p_fluxLM.data(), NVAR*sizeof(REAL*), h2d);
        
        check_cuda_error("alloc_dev_pack_arrays");          
    }
    
    void init_dev_arrays() {
        constexpr auto h2d = hipMemcpyHostToDevice;
        
        hipMemcpy2D(WDr, pNP*sizeof(REAL), host.WDr.data(), NP*sizeof(REAL), NP*sizeof(REAL), NP, h2d);
        hipMemcpy2D(WDs, pNP*sizeof(REAL), host.WDs.data(), NP*sizeof(REAL), NP*sizeof(REAL), NP, h2d);
        hipMemcpy2D(WDt, pNP*sizeof(REAL), host.WDt.data(), NP*sizeof(REAL), NP*sizeof(REAL), NP, h2d);
        hipMemcpy2D(LIFT, pNP*sizeof(REAL), host.LIFT.data(), NP*sizeof(REAL), NP*sizeof(REAL), NFACE*NFP, h2d);
        
        hipMemcpy(rx, host.rx.data(), host.rx.size()*sizeof(REAL), h2d);
        hipMemcpy(ry, host.ry.data(), host.ry.size()*sizeof(REAL), h2d);
        hipMemcpy(rz, host.rz.data(), host.rz.size()*sizeof(REAL), h2d);
        hipMemcpy(sx, host.sx.data(), host.sx.size()*sizeof(REAL), h2d);
        hipMemcpy(sy, host.sy.data(), host.sy.size()*sizeof(REAL), h2d);
        hipMemcpy(sz, host.sz.data(), host.sz.size()*sizeof(REAL), h2d);
        hipMemcpy(tx, host.tx.data(), host.tx.size()*sizeof(REAL), h2d);        
        hipMemcpy(ty, host.ty.data(), host.ty.size()*sizeof(REAL), h2d);
        hipMemcpy(tz, host.tz.data(), host.tz.size()*sizeof(REAL), h2d);
        hipMemcpy(Fscale, host.Fscale.data(), host.Fscale.size()*sizeof(REAL), h2d);
        hipMemcpy(nx, host.nx.data(), host.nx.size()*sizeof(REAL), h2d);
        hipMemcpy(ny, host.ny.data(), host.ny.size()*sizeof(REAL), h2d);
        hipMemcpy(nz, host.nz.data(), host.nz.size()*sizeof(REAL), h2d);
        hipMemcpy(px, host.px.data(), host.px.size()*sizeof(REAL), h2d);
        hipMemcpy(py, host.py.data(), host.py.size()*sizeof(REAL), h2d);
        hipMemcpy(pz, host.pz.data(), host.pz.size()*sizeof(REAL), h2d);
        hipMemcpy(EtoB, host.EtoB.data(), host.EtoB.size()*sizeof(int), h2d);
        hipMemcpy(vmapF, host.vmapF.data(), host.vmapF.size()*sizeof(int), h2d);
        hipMemcpy(vmapP, host.vmapP.data(), host.vmapP.size()*sizeof(int), h2d);
        hipMemcpy(Fmask, host.Fmask.data(), host.Fmask.size()*sizeof(int), h2d);
        hipMemcpy(sendbuf_face_idxs, host.sendbuf_face_idxs.data(), host.sendbuf_face_idxs.size()*sizeof(int), h2d);
        hipMemcpy(sendbuf_nfp_idxs,  host.sendbuf_nfp_idxs.data(),  host.sendbuf_nfp_idxs.size()*sizeof(int), h2d);
        
        check_cuda_error("init_dev_arrays");
    }
    
    void copy_h2d(vector<REAL> &h_src, REAL *d_dst) {
        constexpr auto h2d = hipMemcpyHostToDevice;
        hipMemcpy(d_dst, h_src.data(), h_src.size()*sizeof(REAL), h2d);
        check_cuda_error("copy_h2d");
    }
    
    void copy_d2h(REAL *d_src, vector<REAL> &h_dst) {
        constexpr auto d2h = hipMemcpyDeviceToHost;
        hipMemcpy(h_dst.data(), d_src, h_dst.size()*sizeof(REAL), d2h);
        check_cuda_error("copy_d2h");
    }
        
    void write_u(int tstep) {
        string s = to_string(tstep);
        string fname = "u1_" + string(TDIGIT - s.length(), '0') + s + ".bin";
        
        copy_d2h(p_u[0], u);
        write_bin_file<REAL>(u, host.dataout + fname);
    }    
    
    void free_dev_arrays() {
        hipFree(WDr);
        hipFree(WDs);
        hipFree(WDt);
        hipFree(LIFT);
        hipFree(rx);
        hipFree(ry);
        hipFree(rz);
        hipFree(sx);
        hipFree(sy);
        hipFree(sz);
        hipFree(tx);
        hipFree(ty);
        hipFree(tz);
        hipFree(Fscale);
        hipFree(nx);
        hipFree(ny);
        hipFree(nz);
        hipFree(px);
        hipFree(py);
        hipFree(pz);        
        hipFree(EtoB);
        hipFree(vmapF);
        hipFree(vmapP);        
        hipFree(Fmask);
        hipFree(tau);
        hipFree(maxv);
        hipFree(maxv_fs);
                
        for (int i=0; i<NVAR; i++) {
            hipFree(p_u[i]);
            hipFree(p_k[i]);
            hipFree(p_utmp[i]);
            hipFree(p_ub[i]);            
            hipFree(p_fluxR[i]);
            hipFree(p_fluxS[i]);
            hipFree(p_fluxT[i]);
            hipFree(p_fluxLM[i]);
        }
        
        hipFree(sendbuf_face_idxs);
        hipFree(sendbuf_nfp_idxs);
        hipFree(sendbuf_face);
        hipFree(recvbuf_face);
        hipFree(sendbuf_nfp);
        hipFree(recvbuf_nfp);
    }
};
