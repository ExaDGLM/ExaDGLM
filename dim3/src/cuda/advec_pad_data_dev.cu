#include <vector>
#include <string>
#include <hip/hip_runtime.h>
#include "common.h"

using namespace std;


class DataDev {    
public:
    // DG operator matrix
    REAL *WDr, *WDs, *WDt, *LIFT;
    
    // mesh coordinate transformations
    REAL *rx, *ry, *rz, *sx, *sy, *sz, *tx, *ty, *tz;
    REAL *Fscale;
    
    // mesh normal vectors
    REAL *nx, *ny, *nz;
    
    // mesh physical coordinates
    REAL *px, *py, *pz;
    
    // velocity vectors
    REAL *vx, *vy, *vz;
    
    // DG parameters
    REAL *tau, *maxv, *maxv_fs;
    
    // mesh index arrays
    int *EtoB, *vmapF, *vmapP, *Fmask;
    
    // copy device to host
    vector<REAL> u0;
    
    // governing variable arrays
    REAL *u, *k, *utmp, *ub;
    REAL *fluxR, *fluxS, *fluxT, *fluxLM;
    
    int nelem;
    DataHost host;
    
    
    DataDev(DataHost &host) : host(host) {
        nelem = host.nelem;
        
        // CUDA device
        if (DEVID >= 0) {
            hipError_t err = hipSetDevice(DEVID);
            if (err != hipSuccess)
                printf("Error setting CUDA device: %s\n", hipGetErrorString(err));
        }
        
        // vector arrays
        u0.resize(nelem*NP);
            
        // CUDA arrays
        alloc_dev_arrays();
        init_dev_arrays();
        
        // set the shared memory bank size to 4(float) or 8(double) bytes
        hipSharedMemConfig config = cudaSharedMemBankSizeTByte;
        hipDeviceSetSharedMemConfig(config);        
        check_cuda_error("copy_h2d_all_shared_mem");
    }
    
    ~DataDev() {
        free_dev_arrays();
    }

    void check_cuda_error(string funcname) {
        hipError_t err = hipGetLastError();
        if (err != hipSuccess)
            cout << "CUDA error(" << funcname << "): " << hipGetErrorString(err) << endl;
    }    
    
    void alloc_dev_arrays() {
        hipMalloc(&WDr,  pNP*NP*sizeof(REAL));
        hipMalloc(&WDs,  pNP*NP*sizeof(REAL));
        hipMalloc(&WDt,  pNP*NP*sizeof(REAL));
        hipMalloc(&LIFT, pNP*NFACE*NFP*sizeof(REAL));
        hipMalloc(&rx,     nelem*sizeof(REAL));
        hipMalloc(&ry,     nelem*sizeof(REAL));
        hipMalloc(&rz,     nelem*sizeof(REAL));
        hipMalloc(&sx,     nelem*sizeof(REAL));
        hipMalloc(&sy,     nelem*sizeof(REAL));
        hipMalloc(&sz,     nelem*sizeof(REAL));
        hipMalloc(&tx,     nelem*sizeof(REAL));        
        hipMalloc(&ty,     nelem*sizeof(REAL));        
        hipMalloc(&tz,     nelem*sizeof(REAL));
        hipMalloc(&Fscale, nelem*NFACE*sizeof(REAL));
        hipMalloc(&nx,     nelem*NFACE*sizeof(REAL));
        hipMalloc(&ny,     nelem*NFACE*sizeof(REAL));
        hipMalloc(&nz,     nelem*NFACE*sizeof(REAL));
        hipMalloc(&px,     nelem*NP*sizeof(REAL));
        hipMalloc(&py,     nelem*NP*sizeof(REAL));
        hipMalloc(&pz,     nelem*NP*sizeof(REAL));
        hipMalloc(&vx,     nelem*NP*sizeof(REAL));
        hipMalloc(&vy,     nelem*NP*sizeof(REAL));
        hipMalloc(&vz,     nelem*NP*sizeof(REAL));
        hipMalloc(&EtoB,   nelem*NFACE*sizeof(int));
        hipMalloc(&vmapF,  nelem*NFACE*sizeof(int));
        hipMalloc(&vmapP,  nelem*NFACE*NFP*sizeof(int));
        hipMalloc(&Fmask,  NFACE*NFP*sizeof(int));        
        hipMalloc(&tau,     nelem*NFACE*sizeof(REAL));
        hipMalloc(&maxv,    nelem*NFACE*sizeof(REAL));
        hipMalloc(&maxv_fs, nelem*NFACE*sizeof(REAL));
        hipMalloc(&u,      nelem*NP*sizeof(REAL));
        hipMalloc(&k,      nelem*NP*sizeof(REAL));
        hipMalloc(&ub,     nelem*NFACE*NFP*sizeof(REAL));
        hipMalloc(&utmp,   nelem*pNP*sizeof(REAL));
        hipMalloc(&fluxR,  nelem*pNP*sizeof(REAL));
        hipMalloc(&fluxS,  nelem*pNP*sizeof(REAL));
        hipMalloc(&fluxT,  nelem*pNP*sizeof(REAL));
        hipMalloc(&fluxLM, nelem*pNFF*sizeof(REAL));

        hipMemset(k, 0, nelem*NP*sizeof(REAL));  // zero initialize
        
        check_cuda_error("alloc_dev_arrays");
    }
    
    void init_dev_arrays() {
        constexpr auto h2d = hipMemcpyHostToDevice;
        
        hipMemcpy2D(WDr, pNP*sizeof(REAL), host.WDr.data(), NP*sizeof(REAL), NP*sizeof(REAL), NP, h2d);
        hipMemcpy2D(WDs, pNP*sizeof(REAL), host.WDs.data(), NP*sizeof(REAL), NP*sizeof(REAL), NP, h2d);
        hipMemcpy2D(WDt, pNP*sizeof(REAL), host.WDt.data(), NP*sizeof(REAL), NP*sizeof(REAL), NP, h2d);
        hipMemcpy2D(LIFT, pNP*sizeof(REAL), host.LIFT.data(), NP*sizeof(REAL), NP*sizeof(REAL), NFACE*NFP, h2d);
        
        hipMemcpy(rx, host.rx.data(), host.rx.size()*sizeof(REAL), h2d);
        hipMemcpy(ry, host.ry.data(), host.ry.size()*sizeof(REAL), h2d);
        hipMemcpy(rz, host.rz.data(), host.rz.size()*sizeof(REAL), h2d);
        hipMemcpy(sx, host.sx.data(), host.sx.size()*sizeof(REAL), h2d);
        hipMemcpy(sy, host.sy.data(), host.sy.size()*sizeof(REAL), h2d);
        hipMemcpy(sz, host.sz.data(), host.sz.size()*sizeof(REAL), h2d);
        hipMemcpy(tx, host.tx.data(), host.tx.size()*sizeof(REAL), h2d);        
        hipMemcpy(ty, host.ty.data(), host.ty.size()*sizeof(REAL), h2d);
        hipMemcpy(tz, host.tz.data(), host.tz.size()*sizeof(REAL), h2d);
        hipMemcpy(Fscale, host.Fscale.data(), host.Fscale.size()*sizeof(REAL), h2d);
        hipMemcpy(nx, host.nx.data(), host.nx.size()*sizeof(REAL), h2d);
        hipMemcpy(ny, host.ny.data(), host.ny.size()*sizeof(REAL), h2d);
        hipMemcpy(nz, host.nz.data(), host.nz.size()*sizeof(REAL), h2d);
        hipMemcpy(px, host.px.data(), host.px.size()*sizeof(REAL), h2d);
        hipMemcpy(py, host.py.data(), host.py.size()*sizeof(REAL), h2d);
        hipMemcpy(pz, host.pz.data(), host.pz.size()*sizeof(REAL), h2d);
        hipMemcpy(EtoB, host.EtoB.data(), host.EtoB.size()*sizeof(int), h2d);
        hipMemcpy(vmapF, host.vmapF.data(), host.vmapF.size()*sizeof(int), h2d);
        hipMemcpy(vmapP, host.vmapP.data(), host.vmapP.size()*sizeof(int), h2d);
        hipMemcpy(Fmask, host.Fmask.data(), host.Fmask.size()*sizeof(int), h2d);
        
        check_cuda_error("init_dev_arrays");
    }
    
    void copy_h2d(vector<REAL> &h_src, REAL *d_dst) {
        constexpr auto h2d = hipMemcpyHostToDevice;
        hipMemcpy(d_dst, h_src.data(), h_src.size()*sizeof(REAL), h2d);
        check_cuda_error("copy_h2d");
    }
    
    void copy_d2h(REAL *d_src, vector<REAL> &h_dst) {
        constexpr auto d2h = hipMemcpyDeviceToHost;
        hipMemcpy(h_dst.data(), d_src, h_dst.size()*sizeof(REAL), d2h);
        check_cuda_error("copy_d2h");
    }
        
    void write_u(int tstep) {
        string s = to_string(tstep);
        string fname = "u_" + string(TDIGIT - s.length(), '0') + s + ".bin";
        
        copy_d2h(u, u0);
        write_bin_file<REAL>(u0, host.dataout + fname);
    }    
    
    void free_dev_arrays() {
        hipFree(WDr);
        hipFree(WDs);
        hipFree(WDt);
        hipFree(LIFT);
        hipFree(rx);
        hipFree(ry);
        hipFree(rz);
        hipFree(sx);
        hipFree(sy);
        hipFree(sz);
        hipFree(tx);
        hipFree(ty);
        hipFree(tz);
        hipFree(Fscale);
        hipFree(nx);
        hipFree(ny);
        hipFree(nz);
        hipFree(px);
        hipFree(py);
        hipFree(pz);        
        hipFree(EtoB);
        hipFree(vmapF);
        hipFree(vmapP);        
        hipFree(Fmask);
        hipFree(tau);
        hipFree(maxv);
        hipFree(maxv_fs);
        hipFree(u);
        hipFree(k);
        hipFree(utmp);
        hipFree(ub);            
        hipFree(fluxR);
        hipFree(fluxS);
        hipFree(fluxT);
        hipFree(fluxLM);
    }
};
